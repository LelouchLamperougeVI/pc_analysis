#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

#define bool int
#define true 1
#define false 0

__global__
void tMatch(double const * const templates, double const * const match, double * const results){
  int const tot_i = blockIdx.x * blockDim.x + threadIdx.x;
  // int const block_i = blockDim.x;
  int const thread_i = threadIdx.x;
  int i;

 __shared__ double temp[100];

 temp[thread_i] = match[tot_i];

 __syncthreads();

  for(i = 0; i < 10; i++){
    temp[thread_i] += templates[thread_i];
   __syncthreads();
  }

  results[tot_i] = temp[thread_i];
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
  double const *templates;
  double const *match;
  double *results;
  mxGPUArray const *temp;
  mxGPUArray const *mat;
  mxGPUArray *res;
  mwSize const *dims;

  mxInitGPU();

  temp = mxGPUCreateFromMxArray(prhs[0]);
  templates = (double const *) (mxGPUGetDataReadOnly(temp));
  mat = mxGPUCreateFromMxArray(prhs[1]);
  match = (double const *) (mxGPUGetDataReadOnly(mat));

  dims = mxGPUGetDimensions(mat);
  int const m = dims[0];
  int const n = dims[1];

  res = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(mat), mxGPUGetDimensions(mat), mxGPUGetClassID(mat), mxGPUGetComplexity(mat), MX_GPU_DO_NOT_INITIALIZE);
  results = (double *) (mxGPUGetData(res));

  tMatch<<<n, m>>>(templates, match, results);

  plhs[0] = mxGPUCreateMxArrayOnGPU(res);

  mxGPUDestroyGPUArray(temp);
  mxGPUDestroyGPUArray(mat);
  mxGPUDestroyGPUArray(res);
}
