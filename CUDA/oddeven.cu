#include "hip/hip_runtime.h"
/**
   Odd-even sort algortihm implemented on CUDA for compilation using mexcuda on MATLAB
   Returns matrix with sorted rows and the corresponding indices and ranks
   Usage:
   [sorted, idx, ranks] = oddeven(A);
 *** Input must be converted to 16-bit unsigned integer!

   Tested with MATLAB 2018a, TITAN X, GTX 770, and compiled using CUDA 9.0 and MSVC++ 2015
   Must use with a GPU with compute capability 2.0 and above

   Code is provided "as-is", but feel free to direct complaints to author though the latter reserves the right to ignore them

   Written by HaoRan Chang,
   Polaris Brain Dynamics Research Group,
   Canadian Centre for Behavioural Neuroscience,
   University of Lethbridge, AB, Canada

   Version history:
    2018-07-18: precision of input array reduced to 16-bit unsigned integer;
                can have 24k elements per block/column;
                ergo input matrix from MATLAB must be of type uint16;
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"

#define bool int
#define true 1
#define false 0

#define short uint16_T

#define MAX_BLOCK_SIZE 1024
#define MAX_SHARED 48000

__global__
void populateIdx(short * const idx, int const m){
        int block_i = blockIdx.x;
        int i;

        for(i = 0; i < m; i++)
                idx[block_i*m + i] = i;
}

__global__
void oddevenSort(short const * const x, bool * const sorted, short * const results, short * const idx, int const * n, int const m, int last){
        int const tot_i = blockIdx.x * blockDim.x + threadIdx.x;
        int const block_i = blockIdx.x;
        int const thread_i = threadIdx.x;
        int i;
        short buff;
        bool all_sort = true;

        int m_i = 0;
        for(i = 0; i < thread_i; i++)
                m_i += n[i];

        bool last_thread = last==thread_i;

        extern __shared__ short temp[];

        for(i = 0; i < n[thread_i]; i++)
                temp[m_i + i] = x[block_i * m + m_i + i];

        __syncthreads();

        while(all_sort) {
                all_sort = false;
                sorted[tot_i] = false;
                // __syncthreads();
                for(i = 0; i < n[thread_i] - (n[thread_i] % 2); i+=2) {
                        if(temp[m_i + i] > temp[m_i + i + 1]) {
                                buff = temp[m_i + i];
                                temp[m_i + i] = temp[m_i + i + 1];
                                temp[m_i + i + 1] = buff;

                                buff = idx[block_i*m + m_i + i];
                                idx[block_i*m + m_i + i] = idx[block_i*m + m_i + i + 1];
                                idx[block_i*m + m_i + i + 1] = buff;

                                sorted[tot_i] = true;
                        }
                }
                __syncthreads();

                for(i = 1; i < n[thread_i] - last_thread; i+=2) {
                        if(temp[m_i + i] > temp[m_i + i + 1]) {
                                buff = temp[m_i + i];
                                temp[m_i + i] = temp[m_i + i + 1];
                                temp[m_i + i + 1] = buff;

                                buff = idx[block_i*m + m_i + i];
                                idx[block_i*m + m_i + i] = idx[block_i*m + m_i + i + 1];
                                idx[block_i*m + m_i + i + 1] = buff;

                                sorted[tot_i] = true;
                        }
                }
                __syncthreads();

                for(i = 0; i < blockDim.x; i++)
                        all_sort += sorted[block_i * blockDim.x + i];
        }

        for(i = 0; i < n[thread_i]; i++)
                results[block_i * m + m_i + i] = temp[m_i + i];
}

__global__
void getRanks(short * const results, short * const idx, float * const ranks, int const m){
        int block_i = blockIdx.x + 1;

        int i, buff;
        float count = 0.0f;
        do {
                i = (int) count;
                buff = (int) count;
                while(results[block_i*m - buff - 1] == results[block_i*m - buff - 2] && buff < (m-1))
                        buff++;
                do {
                        ranks[(block_i-1)*m + idx[block_i*m - i - 1]] = (buff - count) / 2.0f + count + 1.0f;
                        i++;
                } while(i <= buff);
                count = buff + 1.0f;
        } while(count < m);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
        if(nrhs != 1)
                mexErrMsgTxt("There needs to be only one input you dummy -_-\"");

        short const *x;
        short *results, *idx;
        float *ranks;
        bool *sorted;
        mxGPUArray const *x_pr, *el_gpu_pr;
        mxGPUArray *sort, *res, *ranks_pr, *idx_pr;
        mxArray *el_pr;
        mwSize const *dims;
        mwSize dimensions[2];
        int i, last;
        int const *el_per_thread_gpu;

        int BLOCK_SIZE;

        mxInitGPU();

        x_pr = mxGPUCreateFromMxArray(prhs[0]);
        if(mxGPUGetClassID(x_pr) != mxUINT16_CLASS)
                mexErrMsgTxt("Input must be of type unint16");
        x = (short const *) (mxGPUGetDataReadOnly(x_pr));

        if(nlhs < 3)
                mexWarnMsgTxt("The whole point of this function is to get ranks in an efficient way. You sure you don't want to assign ranks to an output?");

        dims = mxGPUGetDimensions(x_pr);
        int const m = dims[0];
        int const n = dims[1];

        // bool share = m <= MAX_SHARED/2;
        // if(!share)
        //         mexWarnMsgTxt("The input array is too big to allow shared memory. Using slower global memory scheme.");

        el_pr = mxCreateNumericMatrix(MAX_BLOCK_SIZE, 1, mxINT32_CLASS, mxREAL);
        int *el_per_thread = (int *) mxGetData(el_pr);
        for(i = 0; i < m - 1; i += 2)
                el_per_thread[i/2 % MAX_BLOCK_SIZE] += 2;
        if(m/MAX_BLOCK_SIZE > 0) {
                el_per_thread[MAX_BLOCK_SIZE - 1] += m%2;
                last = MAX_BLOCK_SIZE - 1;
                BLOCK_SIZE = MAX_BLOCK_SIZE;
        }else{
                el_per_thread[m/2 - 1] += m%2;
                last = m/2 - 1;
                BLOCK_SIZE = m/2 + m%2;
        }
        el_gpu_pr = mxGPUCreateFromMxArray(el_pr);
        el_per_thread_gpu = (int const *) mxGPUGetDataReadOnly(el_gpu_pr);

        res = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), dims,
                                  mxUINT16_CLASS, mxGPUGetComplexity(x_pr),
                                  MX_GPU_DO_NOT_INITIALIZE);

        ranks_pr = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), dims,
                                       mxSINGLE_CLASS, mxGPUGetComplexity(x_pr),
                                       MX_GPU_DO_NOT_INITIALIZE);

        idx_pr = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), dims,
                                     mxUINT16_CLASS, mxGPUGetComplexity(x_pr),
                                     MX_GPU_DO_NOT_INITIALIZE);
        idx = (short *) mxGPUGetData(idx_pr);

        populateIdx<<<n, 1>>>(idx, m);

        dimensions[0] = BLOCK_SIZE;
        dimensions[1] = n;
        sort = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), dimensions,
                                   mxINT32_CLASS, mxREAL,
                                   MX_GPU_DO_NOT_INITIALIZE);

        results = (short *) (mxGPUGetData(res));
        ranks = (float *) (mxGPUGetData(ranks_pr));
        sorted = (bool *) (mxGPUGetData(sort));

        oddevenSort<<<n, BLOCK_SIZE, m*sizeof(short)>>>(x, sorted, results, idx, el_per_thread_gpu, m, last);

        getRanks<<<n, 1>>>(results, idx, ranks, m);

        plhs[0] = mxGPUCreateMxArrayOnGPU(res);
        plhs[1] = mxGPUCreateMxArrayOnGPU(idx_pr);
        plhs[2] = mxGPUCreateMxArrayOnGPU(ranks_pr);

        mxGPUDestroyGPUArray(x_pr);
        mxGPUDestroyGPUArray(res);
        mxGPUDestroyGPUArray(sort);
        mxGPUDestroyGPUArray(el_gpu_pr);
        mxGPUDestroyGPUArray(idx_pr);
        mxGPUDestroyGPUArray(ranks_pr);
        mxDestroyArray(el_pr);

        return;
}
