#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"

#define bool int
#define true 1
#define false 0

__global__
void tMatch(double const * const x, bool * const sorted, double * const results, int n){
  int const tot_i = blockIdx.x * blockDim.x + threadIdx.x;
  int const block_i = blockDim.x;
  int const thread_i = threadIdx.x;
  int i;
  double buff;
  bool all_sort = true;

 __shared__ double temp[100];

 for(i = 0; i < n; i++){
   temp[thread_i * n + i] = x[thread_i * n + i];
 }

 __syncthreads();

 while(all_sort){
   all_sort = false;
   sorted[thread_i] = false;
    for(i = 0; i < n; i+=2){
      if(temp[thread_i * n + i] > temp[thread_i * n + i + 2]){
        buff = temp[thread_i * n + i];
        temp[thread_i * n + i] = temp[thread_i * n + i + 2];
        temp[thread_i * n + i + 2] = buff;

        sorted[thread_i] = false;
      }
    for(i = 1; i < n; i+=2){
      if(temp[thread_i * n + i] > temp[thread_i * n + i + 2]){
        buff = temp[thread_i * n + i];
        temp[thread_i * n + i] = temp[thread_i * n + i + 2];
        temp[thread_i * n + i + 2] = buff;

        sorted[thread_i] = false;
      }
     __syncthreads();

     for(i = 0; i < block_i; i++){
       all_sort = all_sort + sorted[i];
     }
    }
  }

  for(i = 0; i < n; i++){
    results[thread_i * n + i] = temp[thread_i * n + i];
  }
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
  double const *x;
  double *sorted, *results;
  bool *sorted
  mxGPUArray const *x_pr;
  mxGPUArray *sort, *res;
  mwSize const *dims;

  mxInitGPU();

  x_pr = mxGPUCreateFromMxArray(prhs[0]);
  x = (double const *) (mxGPUGetDataReadOnly(x_pr));

  dims = mxGPUGetDimensions(x_pr);
  int const m = dims[0];
  int const n = dims[1];

  res = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), mxGPUGetDimensions(x_pr),
                            mxGPUGetClassID(x_pr), mxGPUGetComplexity(x_pr),
                            MX_GPU_DO_NOT_INITIALIZE);

  sort = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), mxGPUGetDimensions(x_pr),
                            mxGPUGetClassID(x_pr), mxGPUGetComplexity(x_pr),
                            MX_GPU_DO_NOT_INITIALIZE);

  results = (double *) (mxGPUGetData(res));
  sorted = (double *) (mxGPUGetData(sort));

  tMatch<<<n, m>>>(templates, match, results);

  plhs[0] = mxGPUCreateMxArrayOnGPU(res);

  mxGPUDestroyGPUArray(temp);
  mxGPUDestroyGPUArray(mat);
  mxGPUDestroyGPUArray(res);
}
