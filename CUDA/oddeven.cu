#include "hip/hip_runtime.h"
/**
   Odd-even sort algortihm implemented on CUDA for compilation using mexcuda on MATLAB
   Returns matrix with sorted rows and the corresponding ranks
   Usage:
   [sorted, ranks] = oddeven(A);

   Tested with MATLAB 2018a, and compiled using CUDA 9.0 and MSVC++ 2015

   Written by HaoRan Chang,
   Polaris Brain Dynamics Research Group,
   Canadian Centre for Behavioural Neuroscience,
   University of Lethbridge, AB, Canada
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "matrix.h"

#define bool int
#define true 1
#define false 0

#define BLOCK_SIZE 1024

__global__
void tMatch(double const * const x, bool * const sorted, double * const results, int const * n, int const m){
        int const tot_i = blockIdx.x * blockDim.x + threadIdx.x;
        int const block_i = blockDim.x;
        int const thread_i = threadIdx.x;
        int i;
        double buff;
        bool all_sort = true;

        __shared__ double temp[100];

        for(i = 0; i < n[thread_i]; i++)
                temp[thread_i * n[thread_i] + i] = x[blockIdx.x * m + thread_i * (n[thread_i] - n[thread_i]%2) + i];

        __syncthreads();

        while(all_sort) {
                all_sort = false;
                sorted[tot_i] = false;
                for(i = 0; i < n[thread_i] - (n[thread_i] % 2); i+=2) {
                        if(temp[thread_i * n[thread_i] + i] > temp[thread_i * n[thread_i] + i + 2]) {
                                buff = temp[thread_i * n[thread_i] + i];
                                temp[thread_i * n[thread_i] + i] = temp[thread_i * n[thread_i] + i + 2];
                                temp[thread_i * n[thread_i] + i + 2] = buff;

                                sorted[tot_i] = false;
                        }
                }
                for(i = 1; i < n[thread_i]; i+=2) {
                        if(temp[thread_i * n[thread_i] + i] > temp[thread_i * n[thread_i] + i + 2]) {
                                buff = temp[thread_i * n[thread_i] + i];
                                temp[thread_i * n[thread_i] + i] = temp[thread_i * n[thread_i] + i + 2];
                                temp[thread_i * n[thread_i] + i + 2] = buff;

                                sorted[tot_i] = false;
                        }
                }
                __syncthreads();

                for(i = 0; i < block_i; i++)
                        all_sort = all_sort + sorted[blockIdx.x * blockDim.x + i];
        }

        for(i = 0; i < n[thread_i]; i++)
                results[blockIdx.x * m + thread_i * (n[thread_i] - n[thread_i]%2) + i] = temp[thread_i * n[thread_i] + i];
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){
        if(nrhs < 1)
                mexErrMsgTxt("There needs to be at least one input you dummy -_-\"");

        double const *x;
        double *results;
        bool *sorted;
        mxGPUArray const *x_pr, *el_gpu_pr;
        mxGPUArray *sort, *res;
        mxArray *el_pr;
        mwSize const *dims;
        mwSize dimensions[2];
        int i;
        int const *el_per_thread_gpu;

        mxInitGPU();

        x_pr = mxGPUCreateFromMxArray(prhs[0]);
        x = (double const *) (mxGPUGetDataReadOnly(x_pr));

        dims = mxGPUGetDimensions(x_pr);
        int const m = dims[0];
        int const n = dims[1];

        el_pr = mxCreateNumericMatrix(BLOCK_SIZE, 1, mxINT32_CLASS, mxREAL);
        int *el_per_thread = (int *) mxGetData(el_pr);
        for(i = 0; i < m - 1; i += 2)
                el_per_thread[i/2 % BLOCK_SIZE] += 2;
        if(m/BLOCK_SIZE > 0) {
                el_per_thread[BLOCK_SIZE - 1] += m%2;
        }else{
                el_per_thread[m/2 - 1] += m%2;
        }
        el_gpu_pr = mxGPUCreateFromMxArray(el_pr);
        el_per_thread_gpu = (int const *) mxGPUGetDataReadOnly(el_gpu_pr);

        for(i=0; i<BLOCK_SIZE; i++)
                mexPrintf("%d ", el_per_thread[i]);

        res = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), dims,
                                  mxGPUGetClassID(x_pr), mxGPUGetComplexity(x_pr),
                                  MX_GPU_DO_NOT_INITIALIZE);

        dimensions[0] = BLOCK_SIZE;
        dimensions[1] = n;
        sort = mxGPUCreateGPUArray(mxGPUGetNumberOfDimensions(x_pr), dimensions,
                                   mxINT32_CLASS, mxREAL,
                                   MX_GPU_DO_NOT_INITIALIZE);

        results = (double *) (mxGPUGetData(res));
        sorted = (bool *) (mxGPUGetData(sort));

        tMatch<<<n, BLOCK_SIZE>>>(x, sorted, results, el_per_thread_gpu, m);

        plhs[0] = mxGPUCreateMxArrayOnGPU(res);

        mxGPUDestroyGPUArray(x_pr);
        mxGPUDestroyGPUArray(res);
        mxGPUDestroyGPUArray(sort);
        mxGPUDestroyGPUArray(el_gpu_pr);
        mxDestroyArray(el_pr);

        return;
}
